#include "hip/hip_runtime.h"
//
// Created by orcun on 25.05.2025.
//

#include "patchwork_gpu/patchwork_gpu.cuh"
#include "cub/device/device_reduce.cuh"
#include "cub/device/device_partition.cuh"

#define NUM_THREADS_PER_PATCH 128
#define WARP_SIZE 32
#define FULL_MASK 0xffffffffu

__device__ __constant__ int cnst_num_sectors_per_ring[256];
__device__ __constant__ std::size_t cnst_num_sectors_per_ring_size;
__device__ __constant__ float cnst_lbr_margin;

__device__ __constant__ double cnst_uprightness_thr;
__device__ __constant__ int cnst_num_rings_of_interest;
__device__ __constant__ double cnst_elevation_thr[64];
__device__ __constant__ double cnst_sensor_height;
__device__ __constant__ double cnst_flatness_thr[64];
__device__ __constant__ bool cnst_using_global_thr;
__device__ __constant__ double cnst_global_elevation_thr;
__device__ __constant__ int cnst_min_num_pts_thr;

template <typename PointT>
void PatchWorkGPU<PointT>::set_cnst_mem()
{
  hipMemcpyToSymbol(HIP_SYMBOL(cnst_lbr_margin), &th_seeds_, sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL(cnst_uprightness_thr), &uprightness_thr_, sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(cnst_num_rings_of_interest), &num_rings_of_interest_, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(cnst_elevation_thr), elevation_thr_.data(), sizeof(double) * elevation_thr_.size());
  hipMemcpyToSymbol(HIP_SYMBOL(cnst_sensor_height), &sensor_height_, sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(cnst_flatness_thr), flatness_thr_.data(), sizeof(double) * flatness_thr_.size());
  hipMemcpyToSymbol(HIP_SYMBOL(cnst_using_global_thr), &using_global_thr_, sizeof(bool));
  hipMemcpyToSymbol(HIP_SYMBOL(cnst_global_elevation_thr), &global_elevation_thr_, sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(cnst_min_num_pts_thr), &num_min_pts_, sizeof(int));
}

// Single kernel version: one block per patch with parallel reduction in shared memory
__global__ void lbr_seed_kernel(
    const float4* patches,
    const uint* num_pts,
    const uint* offsets,
    const double close_zone_z_thresh,
    const int max_ring_first,
    const uint min_pts_thres,
    PointMeta* metas)
{
  const int patch_idx = blockIdx.x;
  const uint n = num_pts[patch_idx];

  if (n == 0) return;
  const bool all_ground = n < min_pts_thres;

  const size_t offset = offsets[patch_idx];
  const bool close_zone = (patch_idx < max_ring_first);

  const int tid = threadIdx.x;
  extern __shared__ float shared_mem[];
  // split shared mem to 2 chunks
  auto* thread_pt_z_sm = shared_mem; // first WARP_SIZE * sizeof(float)
  auto* valid_flags_sm = reinterpret_cast<bool*>(&shared_mem[WARP_SIZE]); // following WARP_SIZE * sizeof(uint)
  valid_flags_sm[tid] = false; // initialize valid flags to false
  __syncthreads();  // make sure every thread sees the cleared flags
  uint warp_cnt = 0;

  //points are already sorted by z in the patch, so first few points in patch must be enough for kernel
  int loop_times = (int)((min_pts_thres + WARP_SIZE-1) / WARP_SIZE);

  for (int iter = 0; iter < loop_times; ++iter) {
    int i = iter * WARP_SIZE + tid;
    if (i < (int)n) {
      float4 pt = patches[offset + i];
      float z   = pt.z;
      if (! valid_flags_sm[tid]) {
        bool flag = (!close_zone) || (z > close_zone_z_thresh);
        warp_cnt += (int)flag;      // accumulate 0 or 1 in this thread’s register
        valid_flags_sm[tid] = flag;
        thread_pt_z_sm[tid] = z;    // store z for this thread
      }
    }
    __syncthreads();

    for (int offset = WARP_SIZE / 2; offset > 0; offset >>= 1) {
      warp_cnt += __shfl_down_sync(0xffffffffu, warp_cnt, offset);
    }

    warp_cnt = __shfl_sync(FULL_MASK, warp_cnt, 0); // broadcast warp count to all threads in the warp

    if (warp_cnt >= min_pts_thres) break;
        // we have more points than needed, sample first min_pts_thres points
      // else; consume rest of the patch, continue to next point or exit loop depending on loop_times
  }

  const uint useful_pts_num = min(warp_cnt, min_pts_thres);

  float thread_pt_z = 0.f;
  if(tid<useful_pts_num)
  {
    thread_pt_z = thread_pt_z_sm[tid];
  }
  __syncthreads();

  for (int offset = WARP_SIZE / 2; offset > 0; offset >>= 1) {
    thread_pt_z += __shfl_down_sync(0xffffffffu, thread_pt_z, offset);
  }

  //broadcast value of thread_pt_z at tid==0 to all threads in the warp
  thread_pt_z = __shfl_sync(FULL_MASK, thread_pt_z, 0);

  const float threshold = (useful_pts_num!=0 ? (thread_pt_z / useful_pts_num) : 0.0f) + cnst_lbr_margin;

  for(unsigned int i = tid; i < n; i += WARP_SIZE){
    // if all_ground is true, we consider all points as ground
    const size_t glob_pt_idx = offset + i;
    metas[glob_pt_idx].ground = all_ground || (patches[glob_pt_idx].z < threshold);
    metas[glob_pt_idx].lbr = threshold; // to be able to visualize the LPR vs chosen points
  }
}

template <typename PointT>
void PatchWorkGPU<PointT>::extract_init_seeds_gpu()
{
  static double lowest_h_margin_in_close_zone =
      (sensor_height_ == 0.0) ? -0.1 : adaptive_seed_selection_margin_ * sensor_height_;

  // for patches in first zone, we only consider the points that are above the sensor height
  // for patches in other zones, all points are used to calculate mean height in patch
  // variable num of threads per patch may be useful.
  dim3 blocks(num_total_sectors_);
  size_t sm_size = WARP_SIZE * (sizeof(float) + sizeof(bool));
  lbr_seed_kernel<<<blocks, WARP_SIZE, sm_size, stream_>>>(
                                                    patches_d,
                                                    num_pts_in_patch_d,
                                                    patch_offsets_d,
                                                    lowest_h_margin_in_close_zone,
                                                    zone_model_->max_ring_index_in_first_zone,
                                                    num_min_pts_,
                                                    metas_d
                                                  );
}

__global__ void compute_patchwise_cov_mat (const float4* patches,
                                            const uint* num_pts_per_patch,
                                            const uint* offsets,
                                            float* cov_out,
                                            PointMeta* metas,
                                            PCAFeature* pca_features
                                          )
{
  static constexpr size_t feat_cnt= 10;
  extern __shared__ double sm_stats[]; // xx, xy, xz, yy, yz, zz, x, y, z count
  const uint patch_idx = blockIdx.x;
  const uint tid = threadIdx.x;
  const uint n = num_pts_per_patch[patch_idx];
  const float4* patch_start = &patches[offsets[patch_idx]];
  const PointMeta* patch_metas = &metas[offsets[patch_idx]];
  float cov_mat[9]; // COL-MAJOR

  #pragma unroll
  for(size_t i=0; i<feat_cnt; ++i) {
    sm_stats[tid * feat_cnt + i] = 0.0;
  }

  __syncthreads();

  double* local_stats = &sm_stats[tid * feat_cnt];

  for (size_t i=tid; i<n; i+=blockDim.x) {
    const bool is_ground = patch_metas[i].ground;
    const float4& pt = patch_start[i];

    local_stats[0] += pt.x * pt.x * is_ground;
    local_stats[1] += pt.x * pt.y * is_ground;
    local_stats[2] += pt.x * pt.z * is_ground;
    local_stats[3] += pt.y * pt.y * is_ground;
    local_stats[4] += pt.y * pt.z * is_ground;
    local_stats[5] += pt.z * pt.z * is_ground;
    local_stats[6] += pt.x * is_ground;
    local_stats[7] += pt.y * is_ground;
    local_stats[8] += pt.z * is_ground;
    local_stats[9] += is_ground;
  }

  __syncthreads(); // local_stats is actuall a part of shared mem.
  for (size_t slice=blockDim.x/2; slice>0; slice>>=1) {
    if (tid < slice) {
      for(size_t stat_idx=0; stat_idx<feat_cnt; ++stat_idx) {
        sm_stats[tid * feat_cnt + stat_idx] += sm_stats[(tid + slice) * feat_cnt + stat_idx];
      }
    }
    __syncthreads();
  }

  if(tid == 0)
  {
    const double count = max(sm_stats[9], 1.0);// avoid division by zero
    const double denom_cov = (count > 1.0) ? (count - 1.0) : 1.0;
    const double denom_mean = (count >= 1.0) ? (count) : 1.0;
    const double inv_count_cov = 1.0 / denom_cov;
    const double inv_count = 1.0 / denom_mean;
    const double x_mean = sm_stats[6] * inv_count;
    const double y_mean = sm_stats[7] * inv_count;
    const double z_mean = sm_stats[8] * inv_count;

    const double xx = (sm_stats[0] - count* x_mean * x_mean) * inv_count_cov;
    const double xy = (sm_stats[1] - count* x_mean * y_mean) * inv_count_cov;
    const double xz = (sm_stats[2] - count* x_mean * z_mean) * inv_count_cov;
    const double yy = (sm_stats[3] - count* y_mean * y_mean) * inv_count_cov;
    const double yz = (sm_stats[4] - count* y_mean * z_mean) * inv_count_cov;
    const double zz = (sm_stats[5] - count* z_mean * z_mean) * inv_count_cov;

    cov_mat[0] = (float)xx; cov_mat[3] = (float)xy; cov_mat[6] = (float)xz;
    cov_mat[1] = (float)xy; cov_mat[4] = (float)yy; cov_mat[7] = (float)yz;
    cov_mat[2] = (float)xz; cov_mat[5] = (float)yz; cov_mat[8] = (float)zz;

    // do not care about patches with insufficient points, we'll handle those later

    #pragma unroll
    for(int i = 0; i < 9; ++i) {
      cov_out[patch_idx*9 + i] = cov_mat[i];
    }
    pca_features[patch_idx].mean_ = make_float3(x_mean, y_mean, z_mean);
  }
}

__global__ void set_patch_pca_features(float* eig_vects,
                                       float* eig_vals,
                                       PCAFeature* pca_features,
                                       const float th_dist)
{
  const uint patch_idx = blockIdx.x;
  PCAFeature pca_feature = pca_features[patch_idx];

  // change the order of eigen values to match OG implementation, just to make it easier to reimplement
  // cusolver sorts them in ascending order.
  pca_feature.singular_values_ = make_float3(static_cast<float>(eig_vals[patch_idx * 3 + 2]),
                                             static_cast<float>(eig_vals[patch_idx * 3 + 1]),
                                             static_cast<float>(eig_vals[patch_idx * 3 ]));

  auto inv_sing_val = 1/ pca_feature.singular_values_.x;
  pca_feature.linearity_ =
      (pca_feature.singular_values_.x - pca_feature.singular_values_.y) * inv_sing_val;
  pca_feature.planarity_ =
      (pca_feature.singular_values_.y - pca_feature.singular_values_.z) * inv_sing_val;

  // 1st vect is the one with least eig val. thus plane normal.
  float* eig_vectors_patch = &eig_vects[patch_idx * 9];
  // eig vectors are stored col-major
  float vx = eig_vectors_patch[0];
  float vy = eig_vectors_patch[1];
  float vz = eig_vectors_patch[2];

  int inv_vect = (vz < 0.0f) ? -1 : 1;
  pca_feature.normal_ = make_float3( vx * inv_vect,
                                     vy * inv_vect,
/* z of normal vector must be pos */ vz * inv_vect);

  pca_feature.d_ = -(pca_feature.normal_.x * pca_feature.mean_.x +
                      pca_feature.normal_.y * pca_feature.mean_.y +
                      pca_feature.normal_.z * pca_feature.mean_.z);

  pca_feature.th_dist_d_ = th_dist - pca_feature.d_;
  pca_features[patch_idx] = pca_feature;
}

__global__ void filter_by_dist2plane(const float4* patches,
                                      const PCAFeature* pca_features,
                                      PointMeta* metas,
                                     const uint num_patched_pts)
{
  const uint tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid>= num_patched_pts) return;

  const PointMeta meta = metas[tid];
  const uint& patch_idx = meta.lin_sec_idx;
  const PCAFeature& feat = pca_features[patch_idx];

  if (meta.iip < 0) return; // if point was previously filtered out, skip it
  const float4& pt = patches[tid];

  const float dist = feat.normal_.x * pt.x +
                      feat.normal_.y * pt.y +
                      feat.normal_.z * pt.z ;

  metas[tid].ground = dist < feat.th_dist_d_;
}


template <typename PointT>
void PatchWorkGPU<PointT>::fit_regionwise_planes_gpu()
// stream selection enforced intentionally, cuSolver is binded with specific stream at somewhere else
{
  static bool work_d_alloced{false};
  static int lwork{0};

  for(size_t i=0; i<num_iter_; ++i) {
    // compute cov matrices for each patch
    const size_t sm_size = NUM_THREADS_PER_PATCH * 10 * sizeof(double);
    compute_patchwise_cov_mat<<<num_total_sectors_, NUM_THREADS_PER_PATCH, sm_size, stream_>>>(
        patches_d, num_pts_in_patch_d, patch_offsets_d, cov_mats_d, metas_d, pca_features_d);

    // run PCA on each patch -> eigenvector w/ least eig, val is the normal
    // cov mat is always positive-semidefinite, so, eigen vectors = singular vectors
    // we can do eigen decomp instead of SVD minor difference from OG implementation

    if(!work_d_alloced)
    {
      // allocate workspace for cuSolver
      hipStreamSynchronize(stream_);
      CUSOLVER_CHECK(hipsolverDnSsyevjBatched_bufferSize(cusolverH,
                                                        HIPSOLVER_EIG_MODE_VECTOR,
                                                        HIPBLAS_FILL_MODE_UPPER,
                                                        3,
                                                        cov_mats_d,
                                                        3,
                                                        eigen_vals_d,
                                                        &lwork,
                                                        syevj_params,
                                                        num_total_sectors_));

      CUDA_CHECK(hipMalloc(reinterpret_cast<void**>(&work_d), sizeof(float) * lwork));
      work_d_alloced = true;
    }

    CUSOLVER_CHECK(hipsolverDnSsyevjBatched(cusolverH,
                                           HIPSOLVER_EIG_MODE_VECTOR,
                                           HIPBLAS_FILL_MODE_UPPER,
                                           3,
                                           cov_mats_d,
                                           3,
                                           eigen_vals_d,
                                           work_d,
                                           lwork,
                                           eig_info_d,
                                           syevj_params,
                                           num_total_sectors_));
    // covariance mats. in cov_mats_d are now eigen vectors

    // compute patchwise PCA features
    set_patch_pca_features<<<num_total_sectors_, 1, 0, stream_>>>(cov_mats_d, eigen_vals_d,
                                                                  pca_features_d, th_dist_);
    //    // choose points by their dist to estimated plane
    dim3 blocks(divup(*num_patched_pts_h, NUM_THREADS_PER_PATCH));
    filter_by_dist2plane<<<blocks, NUM_THREADS_PER_PATCH, 0, stream_>>>(
        patches_d, pca_features_d, metas_d, *num_patched_pts_h);
  }

  auto err = hipGetLastError();
  if (err != hipSuccess)
  {
    std::cerr << "CUDA error in fit_regionwise_planes_gpu: " << hipGetErrorString(err) << std::endl;
  }
}

__device__ PatchState compute_ground_likelihood_estimation_status(
    const int ring_idx,
    const double z_vec,
    const double z_elevation,
    const double surface_variable)
{
  const bool is_too_tilted = z_vec < cnst_uprightness_thr;
  const bool close_ring_flag = (ring_idx < cnst_num_rings_of_interest);
  const bool elev_thr_flag = (z_elevation > -cnst_sensor_height + cnst_elevation_thr[ring_idx]);
  const bool flatness_flag = (cnst_flatness_thr[ring_idx] > surface_variable);
  const bool glob_thr_flag = cnst_using_global_thr && (z_elevation > cnst_global_elevation_thr);

  const bool is_flat_enough = !is_too_tilted && close_ring_flag && elev_thr_flag && flatness_flag;
  const bool is_too_high_elev = !is_too_tilted && close_ring_flag && elev_thr_flag && !flatness_flag;
  const bool is_upright_enough1 = !is_too_tilted && close_ring_flag && !elev_thr_flag;
  const bool is_glob_too_high_elev = !is_too_tilted && !close_ring_flag && glob_thr_flag;
  const bool is_upright_enough2 = !is_too_tilted && !close_ring_flag && !glob_thr_flag;

  // TODO: encode is_*** variables, final output should be single value that'll be both created
  // and used in nondivergent way

  //is_too_tilted, too_high_elevation, patches assumes all points as nonground
  //flat_enough, upright_enough, few_points no overwrite on decided points ground state
}

__global__ void compute_patch_feats(const float4* patches,
                                     const uint* num_pts,
                                     const uint* offsets,
                                     PCAFeature* pca_features,
                                     PointMeta* metas)
{
  // This kernel is not used in the current implementation, but can be used to compute additional patch features
  // if needed in the future.
  const uint patch_idx = blockIdx.x;
  const uint n = num_pts[patch_idx];
  const float4* patch_start = &patches[offsets[patch_idx]];
  PCAFeature& pca_feat = pca_features[patch_idx];
  const float min_singular_val = pca_feat.singular_values_.z;

  const double ground_z_vec = abs(pca_feat.normal_.z);
  const double ground_z_elevation = pca_feat.mean_.z;
  const double surface_variable = min_singular_val /
      (pca_feat.singular_values_.x + pca_feat.singular_values_.y + pca_feat.singular_values_.z);
  auto ring_idx = ring_sec_idx_from_lin_idx(patch_idx).x;

}


template <typename PointT>
void PatchWorkGPU<PointT>::fit_regionwise_planes_gpu()
// stream selection enforced intentionally, cuSolver is binded with specific stream at somewhere else
{
  for(size_t i=0; i<num_iter_; ++i) {
    // compute cov matrices for each patch
    const size_t sm_size = NUM_THREADS_PER_PATCH * 10 * sizeof(double);
    compute_patchwise_cov_mat<<<num_total_sectors_, NUM_THREADS_PER_PATCH, sm_size, stream_>>>(
                                                                                      patches_d,
                                                                                      num_pts_in_patch_d,
                                                                                      patch_offsets_d,
                                                                                      cov_mats_d,
                                                                                      metas_d,
                                                                                      pca_features_d
                                                                                      );
    // run SVD/PCA on each patch -> 3rd eigenvector is the normal
    int lwork{0};
    // cov mat is always positive-semidefinite, eigen vectors = singular vectors
    // we can do eigen decomp instead of SVD
    CUSOLVER_CHECK(hipsolverDnDsyevjBatched_bufferSize(cusolverH,
                                       HIPSOLVER_EIG_MODE_VECTOR,
                                       HIPBLAS_FILL_MODE_LOWER,
                                       3, cov_mats_d,
                                       3, W_solver, &lwork,
                                       syevj_params,
                                       num_total_sectors_)
                    );
    //TODO: check if this can be done once
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&work_d), sizeof(double) * lwork));
    // WARN contents of cov_mats are not guaranteed to be preserved after this call
    CUSOLVER_CHECK(hipsolverDnDsyevjBatched(cusolverH,
                                           HIPSOLVER_EIG_MODE_VECTOR,
                                           HIPBLAS_FILL_MODE_LOWER,
                                           3, cov_mats_d,
                                           3, W_solver,
                                           work_d, lwork,
                                           eig_info_d, syevj_params,
                                           num_total_sectors_)
                   );

    // compute patchwise PCA features
    set_patch_pca_features<<<num_total_sectors_, 1, 0, stream_>>>(
                                                                  cov_mats_d,
                                                                  W_solver,
                                                                  pca_features_d,
                                                                  th_dist_
                                                                 );
    // choose points by their dist to estimated plane
    filter_by_dist2plane<<<num_total_sectors_, NUM_THREADS_PER_PATCH, 0, stream_>>>(
                                                                                  patches_d,
                                                                                  num_pts_in_patch_d,
                                                                                  patch_offsets_d,
                                                                                  pca_features_d,
                                                                                  metas_d
                                                                                  );

    // compute patch features and ground likelihood


    CUDA_CHECK(hipFree(work_d));
  }
}


template class PatchWorkGPU<pcl::PointXYZI>;
template class PatchWorkGPU<PointXYZILID>;
