#include "hip/hip_runtime.h"
//
// Created by orcun on 25.05.2025.
//

#include "patchwork_gpu/patchwork_gpu.cuh"
#define NUM_PATCHES_PER_BLOCK 4
#define NUM_PTS_PER_THREAD 32

__device__ __constant__ int cnst_num_sectors_per_ring[256];
__device__ __constant__ std::size_t cnst_num_sectors_per_ring_size;



__global__ void per_patch_lbr_compute(hipPitchedPtr patches, const uint2 ring_sector_ids,
                                      const float z_thresh, const int num_pts_in_patch,
                                      const bool close_zone, const hipPitchedPtr lbr)
{
  __shared__ float z_mean;
  __shared__ uint valid_pts_in_patch;
  const uint thread_in_patch = threadIdx.x;

  for(uint pt_idx=thread_in_patch ; pt_idx < num_pts_in_patch; pt_idx += NUM_PTS_PER_THREAD)
  {
    // get the point in patch from ring_sector_ids
    const std::size_t row_offset = patches.pitch * patches.ysize * pt_idx +
                              patches.pitch * ring_sector_ids.y;
    const auto* row_ptr = reinterpret_cast<float4*>(static_cast<char*>(patches.ptr) + row_offset);
    float4 pt = row_ptr[ring_sector_ids.x];
    if (close_zone) // TODO shouldnt cause warp divergence but recheck and ensure
    {
      if (pt.z > z_thresh) {
        atomicAdd(&z_mean, pt.z);
        atomicAdd(&valid_pts_in_patch, 1);
      }
    } else {
      atomicAdd(&z_mean, pt.z);
      atomicAdd(&valid_pts_in_patch, 1);
    }
  }
  __syncthreads(); // sync all threads in patch
  if (thread_in_patch == 0) {
      // compute the mean height of the patch
      const bool empty_patch = (valid_pts_in_patch == 0);
      const float tmp_norm = empty_patch > 0 ? static_cast<float>(valid_pts_in_patch) : 1.0f; // avoid division by zero
      z_mean = z_mean / tmp_norm;

      auto lbr_row = reinterpret_cast<float*>( static_cast<char*>(lbr.ptr) +
                                                 lbr.pitch * ring_sector_ids.y );
      lbr_row[ring_sector_ids.x] = empty_patch ? 0.0f : z_mean;
  }
}

__global__ void per_patch_manager_kernel(const hipPitchedPtr patches_ptr,
                                          const hipPitchedPtr num_pts_in_patch_ptr,
                                          const float z_thres,
                                          const int max_ring_idx_in_first_zone,
                                          const uint min_pts_thres,
                                         hipPitchedPtr lbr_d)
{
//  const int* num_sectors_per_ring, set as consant
  const uint ring_idx = threadIdx.x;
  const uint sector_idx = threadIdx.y;
  const uint2 ring_sector_ids = make_uint2(ring_idx, sector_idx);

  // some sectors actually does not exist, they're just here to keep a 3d data structure
  const bool dummy_sector = sector_idx >= cnst_num_sectors_per_ring[ring_idx];

  auto* num_ptr = reinterpret_cast<int*>(
                          static_cast<char*>(num_pts_in_patch_ptr.ptr) +
                                num_pts_in_patch_ptr.pitch * sector_idx
                          ) + ring_idx;
  if (dummy_sector)
  {
    *num_ptr = 0; // no points in this sector
    return;
  }
  const int num_pts_in_patch = *num_ptr;
  const bool is_close = (ring_idx <= max_ring_idx_in_first_zone);
  const bool few_points = (num_pts_in_patch < min_pts_thres);

  if(few_points) return;
  // TODO do not forget about few_points case
  dim3 threads( divup(num_pts_in_patch, NUM_PTS_PER_THREAD) );

  per_patch_lbr_compute<<<threads, 1>>>(patches_ptr, ring_sector_ids, z_thres,
                                        num_pts_in_patch, is_close, lbr_d);

}

template <typename PointT>
void PatchWorkGPU<PointT>::launch_seed_extract_kernel(hipStream_t& stream)
{
  static double lowest_h_margin_in_close_zone =
      (sensor_height_ == 0.0) ? -0.1 : adaptive_seed_selection_margin_ * sensor_height_;

  // for patches in first zone, we only consider the points that are above the sensor height
  // for patches in other zones, all points are used to calculate mean height in patch
  // variable num of threads per patch may be useful.
  dim3 threads(zone_model_->num_total_rings_, zone_model_->max_num_sectors_);
  per_patch_manager_kernel<<<threads, 1, 0, stream>>>(
      patches_d, num_pts_in_patch_d, lowest_h_margin_in_close_zone,
      zone_model_->max_ring_idx_in_first_zone_, num_min_pts_);

}