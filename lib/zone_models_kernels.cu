#include "hip/hip_runtime.h"
//
// Created by orcun on 10.05.2025.
//
#include <hipcub/hipcub.hpp>
#include "patchwork_gpu/zone_models_gpu.cuh"

// __device__ functions are inlined by default

__device__ __constant__ float cnst_sqr_boundary_ranges[256];
__device__ __constant__ std::size_t cnst_boundary_ranges_size;
__device__ __constant__ float cnst_sqr_max_range;
__device__ __constant__ int cnst_num_sectors_per_ring[256];
__device__ __constant__ std::size_t cnst_num_sectors_per_ring_size;


__device__ float xy2sqr_r(const float &x, const float &y) { return x * x + y * y; }

__device__ float xy2theta(const float &x, const float &y) {  // 0 ~ 2 * PI
  auto atan_value = atan2f(y, x);       // EDITED!
  return atan_value > 0 ? atan_value : atan_value + 2 * M_PI;  // EDITED!
}

__device__ int get_ring_idx(const float &x, const float &y) {
  float sqr_r = xy2sqr_r(x, y);
  // Exception for UAVs such as NTU VIRAL dataset
  if (sqr_r < cnst_sqr_boundary_ranges[0]) {
    return INVALID_RING_IDX;
  }
  if (sqr_r > cnst_sqr_max_range) {
    return OVERFLOWED_IDX;
  }

  for (int i = 1; i < cnst_boundary_ranges_size; ++i) {
    if (sqr_r <= cnst_sqr_boundary_ranges[i]) {
      return i-1;
    }
  }
}

__device__ int get_sector_idx(const float &x, const float &y, const int ring_idx) {
  float theta = xy2theta(x, y);
  int num_sectors = cnst_num_sectors_per_ring[ring_idx];
  float sector_size = 2.0 * M_PI / static_cast<float>(num_sectors);

  // min: for defensive programming
  return min(static_cast<int>(theta / sector_size), num_sectors - 1);
}

__device__ int2 get_ring_sector_idx(const float &x, const float &y)
{
  int ring_idx = get_ring_idx(x, y);
  if (ring_idx == INVALID_RING_IDX || ring_idx == OVERFLOWED_IDX) {
    return make_int2(ring_idx, ring_idx);
  }

  int sector_idx = get_sector_idx(x, y, ring_idx);
  return make_int2(ring_idx, sector_idx);
}

template<typename PointT>
__global__ void count_patches_kernel( PointT *points,
                                      uint* num_pts_in_patch,
                                      PointMeta* in_metas,
                                      float z_thresh,
                                      int num_pts_in_cloud)
{
  std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= num_pts_in_cloud) return;

  const PointT &pt = points[idx];
  int2 ring_sector_indices = get_ring_sector_idx(pt.x, pt.y);

  const size_t lin_sector_idx = resolve_lin_sec_idx(ring_sector_indices.x, ring_sector_indices.y);
  uint* patch_numel_ptr = num_pts_in_patch + lin_sector_idx;
  int iip = -1; // intra-patch index

  if (ring_sector_indices.x >= 0 && pt.z > z_thresh)
  {
   iip = atomicAdd(patch_numel_ptr, 1); // save this as idx in patch
  }

  in_metas[idx] = make_point_meta( ring_sector_indices.x,
                               ring_sector_indices.y,
                               lin_sector_idx,
                               iip);
}

template<typename PointT>
__global__ void move_points_to_patch_kernel(PointT* points,
                                            const PointMeta* in_metas_d,
                                            PointMeta* metas_d,
                                            float* z_keys,
                                            const uint* offsets_d,
                                            PointT* patches_d, float z_thresh,
                                            uint num_pc_points) {
  std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= num_pc_points) return;

  const PointT &pt = points[idx];
//  if (pt.z < z_thresh) return; //  this is obsolete. meta.iip already validates points
  PointMeta meta = in_metas_d[idx];
  if (meta.iip == -1) return;
  const auto pt_offset = offsets_d[meta.lin_sec_idx] + meta.iip;
  patches_d[pt_offset] = pt;
  metas_d[pt_offset] = meta;
  z_keys[pt_offset] = pt.z;
}

template<typename PointT>
__global__ void apply_permutation_kernel(const PointT* in_patches_d,
                                          const PointMeta* in_metas_d,
                                          PointT* sorted_patches_d,
                                          PointMeta* sorted_metas_d,
                                          const uint* permutation,
                                          uint num_pc_points) {
  const uint dst_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (dst_idx >= num_pc_points) return;
  const uint src_idx = permutation[dst_idx];

  const PointMeta& meta = in_metas_d[src_idx];
  if (meta.iip == -1) return;
  const PointT& pt = in_patches_d[src_idx];
  sorted_patches_d[dst_idx] = pt;
  sorted_metas_d[dst_idx] = meta;
}


template<typename PointT>
bool ConcentricZoneModelGPU<PointT>::create_patches_gpu(PointT* cloud_in_d, int num_pc_pts,
                                                          uint* num_pts_in_patch_d,
                                                          PointMeta* in_metas_d,
                                                          PointMeta* metas_d,
                                                          uint* offsets_d,
                                                          uint num_total_sectors,
                                                          PointT* patches_d,
                                                          uint& num_patched_pts_h,
                                                          float* sorted_z_d,
                                                          hipStream_t& stream)
{
  if (num_pc_pts > max_num_pts) {
    throw std::runtime_error("Number of points in the point cloud exceeds the maximum limit.");
  }

  float z_thresh = -sensor_height_ - 2.0; // threshold for z coordinate

  static const uint num_threads = 512;
  dim3 threads(num_threads);
  dim3 blocks(divup(num_pc_pts, num_threads));

  if (cub_dev_scan_sum_tmp_ != nullptr) {
    // this scratch memory must be replaced every time since num points is not consistent
    hipFree(cub_dev_scan_sum_tmp_);
    cub_dev_scan_sum_tmp_ = nullptr;
  }
  // compute the num of points in each patch
  count_patches_kernel<<<blocks, threads, 0, stream>>>(cloud_in_d,
                                                        num_pts_in_patch_d,
                                                        in_metas_d,
                                                        z_thresh,
                                                        num_pc_pts);

  // compute patch offsets
  // query the temporary storage size for the exclusive sum
  static size_t sum_query_bytes{0};
  CUDA_CHECK( hipcub::DeviceScan::InclusiveSum(
                      /* d_temp_storage */ nullptr,
                      /* temp_storage_bytes */ sum_query_bytes,
                      /* d_in */ num_pts_in_patch_d,
                      /* d_out */ offsets_d+1,
                      /* num_items */ num_total_sectors,
                      /* stream */ stream)
  );
  // to have both inclusive and exclusive sum, we have num_segments+1 buffer. starting to write 1st
//  element (instead of 0th) we keep the first patch offset as zero, but also keep the correct buffer end
//  at num_segments+1th element. End offset is specifically needed for segmented sort
  if (cub_dev_scan_sum_tmp_ == nullptr
      || sum_query_bytes > cub_dev_scan_sum_tmp_bytes) {

    if (cub_dev_scan_sum_tmp_) {
      CUDA_CHECK(hipFreeAsync(cub_dev_scan_sum_tmp_, stream));
    }
    // Allocate exactly what CUB needs
    cub_dev_scan_sum_tmp_bytes = sum_query_bytes;
    CUDA_CHECK(hipMallocAsync(&cub_dev_scan_sum_tmp_, cub_dev_scan_sum_tmp_bytes, stream));
  }

  CUDA_CHECK( hipcub::DeviceScan::InclusiveSum(
                  /* d_temp_storage */    cub_dev_scan_sum_tmp_,
                  /* temp_storage_bytes */ cub_dev_scan_sum_tmp_bytes,
                  /* d_in */              num_pts_in_patch_d,
                  /* d_out */             offsets_d+1,
                  /* num_items */         num_total_sectors,
                  /* stream */            stream
              ));
  hipStreamSynchronize(stream); // end compute offsets
  CUDA_CHECK(hipGetLastError());

  hipMemcpyAsync(num_pts_per_patch_h.data(), num_pts_in_patch_d,
                  sizeof(uint) * num_total_sectors, hipMemcpyDeviceToHost, czm_stream_);

  dim3 move_threads(num_threads);
  dim3 move_blocks(divup(num_pc_pts, num_threads));
  // move points from input cloud to patches buffer
  move_points_to_patch_kernel<<<move_blocks, move_threads,0, stream>>>(cloud_in_d,
                                                                        in_metas_d,
                                                                        metas_interm,
                                                                        z_keys_d_,
                                                                        offsets_d, unsorted_patches_d_,
                                                                        z_thresh, num_pc_pts);
  hipStreamSynchronize(czm_stream_);
  num_patched_pts_h = std::accumulate(num_pts_per_patch_h.begin(), num_pts_per_patch_h.end(), 0u);

  // sort workspace size just depends on num_total_sectors,
  //  thus the workspace can be reused through the lifetime of the program
  static size_t sort_query_bytes = 0;
  hipcub::DeviceSegmentedSort::SortPairs(
                                      nullptr, sort_query_bytes,
                                      z_keys_d_, sorted_z_d,
                                      raw_perm_idx, sorted_perm_idx,
                                      num_patched_pts_h, num_total_sectors,
                                      offsets_d, offsets_d + 1, stream);

  if (cub_sort_tmp_d == nullptr || sort_query_bytes > cub_sort_tmp_bytes)
  {
    // if prev scratch pad allocation is not enough, free and realloc
    if (cub_sort_tmp_d) hipFreeAsync(cub_sort_tmp_d, stream);
    cub_sort_tmp_bytes = sort_query_bytes;
    // Allocate temporary storage
    hipMallocAsync(&cub_sort_tmp_d, cub_sort_tmp_bytes, stream);
  }
  // sort pts within patches by z. only get idx mapping
  hipcub::DeviceSegmentedSort::SortPairs(
      cub_sort_tmp_d, cub_sort_tmp_bytes,
      z_keys_d_, sorted_z_d, raw_perm_idx, sorted_perm_idx,
      num_patched_pts_h, num_total_sectors, offsets_d, offsets_d + 1, stream);

  dim3 perm_threads(num_threads);
  dim3 perm_blocks(divup(num_patched_pts_h, num_threads));
  apply_permutation_kernel<<<perm_blocks, perm_threads, 0, stream>>>(unsorted_patches_d_, metas_interm,
                                                                     patches_d, metas_d,
                                                                     sorted_perm_idx, num_patched_pts_h
                                                                     );

  return true;
}

__global__ void set_lin_inc_mem(uint* dst, const uint numel)
{
  const uint tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid>=numel) return;
  dst[tid] = tid;
}

template<typename PointT>
void ConcentricZoneModelGPU<PointT>::set_permute_idx()
{
  dim3 threads(512);
  dim3 blocks(divup(max_num_pts, 512));

  set_lin_inc_mem<<<blocks, threads, 0, czm_stream_>>>(raw_perm_idx, max_num_pts);
}

template<typename PointT>
void ConcentricZoneModelGPU<PointT>::set_cnst_mem()
{
  // we declared __constant__ symbols as extern in cuh but didnt  defined them.
  // also only compiled this file, so symbols wouldnt be compiled if they didn't reside in cpp/cu file
  // as a result this function or at least one reference of each symbol must have been in cu/cpp file
  // that's why just this func is here cu file.
  CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(cnst_sqr_boundary_ranges), sqr_boundary_ranges_.data(),
                                sizeof(float) * sqr_boundary_ranges_.size()));
  auto tmp = sqr_boundary_ranges_.size();
  CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(cnst_boundary_ranges_size), &tmp, sizeof(std::size_t)));
  CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(cnst_sqr_max_range), &sqr_max_range_, sizeof(float)));
  CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(cnst_num_sectors_per_ring), num_sectors_per_ring_.data(),
                                sizeof(int) * num_sectors_per_ring_.size()));
  auto tmp2 = num_sectors_per_ring_.size();
  CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(cnst_num_sectors_per_ring_size), &tmp2,sizeof(std::size_t)));
}

template class ConcentricZoneModelGPU<PointXYZILID>;

